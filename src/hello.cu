
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void cuda_hello()
{
    printf("Hello World from GPU And Docker!\n");
}

int main()
{
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();
    return 0;
}